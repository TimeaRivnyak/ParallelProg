#include "hip/hip_runtime.h"
/* Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <iostream>

// This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void my_stencil_kernel(double *A, double *Anew, int imax, int jmax, double *d_x)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int id = threadIdx.x + threadIdx.y * blockDim.x;
    __shared__ double local[64];
    if (i >= 1 && i < imax + 1 && j >= 1 && j < jmax + 1)
    {
        if (id < (imax * jmax)){
            local[threadIdx.x + threadIdx.y] = A[id];
        }
        else{
            local[threadIdx.x + threadIdx.y] = 0.0;
        }
        Anew[(j) * (imax + 2) + i] = 0.25f * (A[(j) * (imax + 2) + i + 1] + A[(j) * (imax + 2) + i - 1] + A[(j - 1) * (imax + 2) + i] + A[(j + 1) * (imax + 2) + i]);
        local[threadIdx.x + threadIdx.y] = Anew[id];
        for (int d = blockDim.x * blockDim.y >> 1; d >= 1; d >>= 1)
        {
            __syncthreads();
            // (j) * (imax + 2) + i, fabs(Anew[(j) * (imax + 2) + i] - A[(j) * (imax + 2) + i])
            if ((threadIdx.x + threadIdx.y) < d) local[threadIdx.x + threadIdx.y] = fabs(local[threadIdx.x + threadIdx.y+d] - local[threadIdx.x + threadIdx.y+d]);
        }

        if (threadIdx.x == 0)
        {
            atomicMax(d_x, local[0]);
        }
    }
}

__global__ void my_copy_kernel(double *A, double *Anew, int imax, int jmax)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= 1 && i < imax + 1 && j >= 1 && j < jmax + 1)
        A[(j) * (imax + 2) + i] = Anew[(j) * (imax + 2) + i];
}

int main(int argc, char **argv)
{
    // Size along y
    int jmax = 4094;
    // Size along x
    int imax = 4094;
    int iter_max = 1000;

    const double pi = 2.0 * asin(1.0);
    const double tol = 1.0e-5;
    double error = 1.0;

    double *A;
    double *Anew;
    double *y0;

    A = (double *)malloc((imax + 2) * (jmax + 2) * sizeof(double));
    Anew = (double *)malloc((imax + 2) * (jmax + 2) * sizeof(double));
    y0 = (double *)malloc((imax + 2) * sizeof(double));

    memset(A, 0, (imax + 2) * (jmax + 2) * sizeof(double));

    // set boundary conditions
    for (int i = 0; i < imax + 2; i++)
        A[(0) * (imax + 2) + i] = 0.0;

    for (int i = 0; i < imax + 2; i++)
        A[(jmax + 1) * (imax + 2) + i] = 0.0;

    for (int j = 0; j < jmax + 2; j++)
    {
        y0[j] = sin(pi * j / (jmax + 1));
        A[(j) * (imax + 2) + 0] = y0[j];
    }

    for (int j = 0; j < imax + 2; j++)
    {
        y0[j] = sin(pi * j / (jmax + 1));
        A[(j) * (imax + 2) + imax + 1] = y0[j] * exp(-pi);
    }

    printf("Jacobi relaxation Calculation: %d x %d mesh\n", imax + 2, jmax + 2);

    // double t1 = omp_get_wtime();
    int iter = 0;

    for (int i = 1; i < imax + 2; i++)
        Anew[(0) * (imax + 2) + i] = 0.0;

    for (int i = 1; i < imax + 2; i++)
        Anew[(jmax + 1) * (imax + 2) + i] = 0.0;

    for (int j = 1; j < jmax + 2; j++)
        Anew[(j) * (imax + 2) + 0] = y0[j];

    for (int j = 1; j < jmax + 2; j++)
        Anew[(j) * (imax + 2) + jmax + 1] = y0[j] * expf(-pi);

    double *d_A;
    double *d_Anew;
    double *d_x;
    cudaCheck(hipMalloc(&d_A, (imax + 2) * (jmax + 2) * sizeof(double)));
    cudaCheck(hipMemcpy(d_A, A, (imax + 2) * (jmax + 2) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&d_Anew, (imax + 2) * (jmax + 2) * sizeof(double)));
    cudaCheck(hipMemcpy(d_Anew, Anew, (imax + 2) * (jmax + 2) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&d_x, (imax + 2) * (jmax + 2) * sizeof(double)));
    cudaCheck(hipMemset(d_x, 0, (imax + 2) * (jmax + 2) * sizeof(double)));

    while (error > tol && iter < iter_max)
    {
        error = 0.0;

        dim3 block(16, 4);
        dim3 grid((imax + 2 - 1) / 16 + 1, (jmax + 2 - 1) / 4 + 1);
        cudaCheck(hipDeviceSynchronize());
        my_stencil_kernel<<<grid, block>>>(d_A, d_Anew, imax, jmax, d_x);
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipMemcpy(&error, d_x, sizeof(double), hipMemcpyDeviceToHost));
        // No stencil accesses to Anew, no halo exchange necessary
        my_copy_kernel<<<grid, block>>>(d_A, d_Anew, imax, jmax);
        if (iter % 100 == 0)
            printf("%5d, %0.6f\n", iter, error);

        iter++;
    }

    // double runtime = omp_get_wtime()-t1;
    double runtime = 1;
    printf(" total: %f s\n", runtime);
}
